/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define BLOCK_SIZE 1024        // Define by myself

void check_param(void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		        /* total points along string */
    rcode;                  	/* generic return code */
    
float  values[MAXPOINTS+2], 	/* values at time t */ 
       *valuesInThread;       /* values in threads */

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

__global__ void run_parallel(float *valuesInThread, int tpoints, int nsteps)
{
    int i, k;
    float x, fac, tmp;
    float dtime, c, dx, tau, sqtau;
    float value, new_val, old_val;

    /* init_line() */
    fac = 2.0 * PI;
    k = 1 + blockIdx.x * BLOCK_SIZE + threadIdx.x;
    tmp = tpoints - 1;
    x = (k - 1) / tmp;
    value = sin (fac * x);
    old_val = value;

    /* do_math() */
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    /* update() */
    if(k <= tpoints) {
      for (i = 1; i<= nsteps; i++) {
        if ((k == 1) || (k  == tpoints))
          new_val = 0.0;
        else
          new_val = (2.0 * value) - old_val + (sqtau * -2.0 * value);
        /* Update old values with new values */
        old_val = value;
        value = new_val;
      }
      /* Copy to value in threads */
      valuesInThread[k] = value;
    }
}


/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
  int size;
  int block_num;
  
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
 
  size = (tpoints + 1) * sizeof(float);
  hipMalloc((void**) &valuesInThread, size);
  
	printf("Initializing points on the line...\n");
  
	printf("Updating all points for all time steps...\n");
 
  if (tpoints % BLOCK_SIZE == 0) {
    block_num = tpoints / BLOCK_SIZE;
  }
  else
    block_num = tpoints / BLOCK_SIZE + 1;
  run_parallel<<<block_num, BLOCK_SIZE>>>(valuesInThread, tpoints, nsteps);
  
  hipMemcpy(values, valuesInThread, size, hipMemcpyDeviceToHost);
  hipFree(valuesInThread);
  
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}